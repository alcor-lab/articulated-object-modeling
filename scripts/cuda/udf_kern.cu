// Authors: Valsamis Ntouskos, ntouskos@diag.uniroma1.it; Bruno Cafaro, cafaro@diag.uniroma1.it 


#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

#define sq(x) ((x)*(x))

__global__ void udf(float *udf, float *ipf, unsigned int *ind, 
                    const float *vert3, const float *normals3,
                    float *min3, float *del3, unsigned int *res3, unsigned int numpts)
{
	unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i>=res3[0]) {
        return;
    }
	
    unsigned int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (j>=res3[1]) {
        return;
    }
    
	unsigned int k = threadIdx.z + blockIdx.z * blockDim.z;
    if (k>=res3[2]) {
        return;
    }
    
	unsigned int voxind = i+j*res3[0]+k*res3[0]*res3[1];

	float xvox = min3[0] + del3[0]*(i+0.5);
    float yvox = min3[1] + del3[1]*(j+0.5);
    float zvox = min3[2] + del3[2]*(k+0.5);
    
    for( unsigned int kk = 0; kk < numpts; kk++ ) {
        float xvert = vert3[kk]; 
        float yvert = vert3[numpts*1+kk]; 
        float zvert = vert3[numpts*2+kk];
        
        float dis = sqrt(sq(xvox-xvert)+sq(yvox-yvert)+sq(zvox-zvert));
        
        if (dis < udf[voxind]){
            udf[voxind] = dis;
            ind[voxind] = kk;
            
            float xnorm = normals3[kk];
            float ynorm = normals3[numpts*1+kk];
            float znorm = normals3[numpts*2+kk];
            
            ipf[voxind] = - ((xvox - xvert)*xnorm + (yvox - yvert)*ynorm + (zvox - zvert)*znorm);
        }
    }
	//__syncthreads();
}
